
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

/* vectorAdd version 1 */

__global__ void vectorAdd(int* a, int* b, int* c)
{
		*c = *a + *b;
}

int main(void) {
		int size  = sizeof(int);

		int h_A,h_B;
		scanf("%d %d",&h_A,&h_B);
		int h_C;

		int *d_A = NULL;
		int *d_B = NULL;
		int *d_C = NULL;
		hipMalloc((void **)&d_A,size);
		hipMalloc((void **)&d_B,size);
		hipMalloc((void **)&d_C,size); 


		hipMemcpy(d_A, &h_A, size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, &h_B, size, hipMemcpyHostToDevice);

		clock_t begin = clock();
		vectorAdd<<<4,1>>> (d_A,d_B,d_C);
		hipMemcpy(&h_C, d_C, size, hipMemcpyDeviceToHost);
		clock_t end = clock();
		if (h_C == h_A + h_B)
		{
				printf("add correction! c is %d\n",h_C);
		}

		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);

		printf("time: %ld\n",end-begin);

		printf("Done\n");
		return 0;
}
