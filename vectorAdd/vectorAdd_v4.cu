
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 512
#define THREADS_PER_BLCOK 512

/* vectorAdd version 4 */

__global__ void vectorAdd(int* a, int* b, int* c)
{
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		c[index] = a[index] + b[index];
}



int main(void) {
	int size  = sizeof(int)*N;
	
	int *h_A,*h_B;
	int *h_C;

	h_A = (int *)malloc(size);
	h_B = (int *)malloc(size);
	h_C = (int *)malloc(size); 

	scanf("%d %d",h_A,h_B);

	
	int *d_A = NULL;
	int *d_B = NULL;
	int *d_C = NULL;

	hipMalloc((void **)&d_A,size);
	hipMalloc((void **)&d_B,size);
	hipMalloc((void **)&d_C,size); 
	
	
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	clock_t begin = clock();
	vectorAdd<<<N/THREADS_PER_BLCOK,THREADS_PER_BLCOK>>> (d_A,d_B,d_C);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	clock_t end = clock();

	if (*h_C == *h_A + *h_B)
	{
			printf("add correction! c is %d\n",*h_C);
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	printf("time: %ld\n",end-begin);
	printf("Done\n");
	return 0;
}
