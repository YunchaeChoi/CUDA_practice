
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

/* vectorAdd version 2 */

__global__ void vectorAdd(int* a, int* b, int* c)
{
		c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512

int main(void) {
	int size  = sizeof(int)*N;
	
	int *h_A,*h_B;
	int *h_C;

	h_A = (int *)malloc(size);
	h_B = (int *)malloc(size);
	h_C = (int *)malloc(size); 

	scanf("%d %d",h_A,h_B);

	
	int *d_A = NULL;
	int *d_B = NULL;
	int *d_C = NULL;

	hipMalloc((void **)&d_A,size);
	hipMalloc((void **)&d_B,size);
	hipMalloc((void **)&d_C,size); 
	
	
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	clock_t begin = clock();
	vectorAdd<<<1,N>>> (d_A,d_B,d_C);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	clock_t end = clock();

	if (*h_C == *h_A + *h_B)
	{
			printf("add correction! c is %d\n",*h_C);
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	printf("time: %ld\n",end-begin);
	printf("Done\n");
	return 0;
}
